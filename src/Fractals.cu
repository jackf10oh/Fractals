#include "hip/hip_runtime.h"
#ifndef MATRIX_CU
#define MATRIX_CU

#include "Fractals.hpp"
#include<math.h>

inline void checkLastError()
{
  hipError_t err;
  err=hipGetLastError();
  if(err!=hipSuccess) printf("error from cuda:%s\n", hipGetErrorString(err));
};

template<complexFunc_t F>
__global__ void iteration_kernel(int* iter_array, int max_iters, double radius, Complex* val_array, int num_elems)
{
  int stride = gridDim.x * blockDim.x;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for(int i=idx; i<num_elems; i+=stride)
  {
    iter_array[i]=0;

    Complex c_init=val_array[i], z=val_array[i]; // complex numbers c,z = a+ib
    int iter=1; // leave iter_array as 0 if recursion doesn't explode.
    do
    {
      // if current val > radius of fractal
      if(modulus2(z) >= (radius*radius)) 
      {
        val_array[i]=z; // copy z back to val_array
        iter_array[i]=iter; // update iter_array to nonzero interger
        break;
      }; 
      z = (*F)(c_init,z); // assignment handled in my_recursive
      ++iter;
    }
    while(iter<max_iters); // loop while inside radius and iterations remain
  }
};



//constructors
template<complexFunc_t F>
Fractal<F>::Fractal() //default
{
  num_iters=9999;
  nRows=1020; nCols=1980; 

  x1=-2; y1=2;
  x2=2; y2=-2;

  // allocate memory
  val_array = vector<vector<Complex>>(nRows, vector<Complex>(nCols));
  iter_array = vector<vector<int>>(nRows, vector<int>(nCols,0));

  // fill values
  for(int i=0; i<nRows; i++)
  {
    double y = y1 + (y2-y1)*(double(i)/(nRows-1));
    for(int j=0; j<nCols; j++)
    {
      double x = x1 + (x2-x1)*(double(j)/(nCols-1));
      val_array[i][j] = Complex(x,y);
    }
  }
};

template<complexFunc_t F>
Fractal<F>::Fractal(Complex point1, Complex point2, tuple<int,int> dims, int num_iters_init) // from dims + n_iters
{
  num_iters=num_iters_init;

  nRows=get<0>(dims); nCols=get<1>(dims); 

  x1=point1.real; y1=point1.im;
  x2=point2.real; y2=point2.im;

  // allocate memory
  val_array = vector<vector<Complex>>(nRows, vector<Complex>(nCols));
  iter_array = vector<vector<int>>(nRows, vector<int>(nCols,0));

  // fill values
  for(int i=0; i<nRows; i++)
  {
    double y = y1 + (y2-y1)*(double(i)/(nRows-1));
    for(int j=0; j<nCols; j++)
    {
      double x = x1 + (x2-x1)*(double(j)/(nCols-1));
      val_array[i][j] = Complex(x,y);
    }
  }
};

template<complexFunc_t F>
Fractal<F>::Fractal(const Fractal &source) // copy 
{
  num_iters=source.num_iters;
  nRows=source.nRows; nCols=source.nCols; 

  x1=source.x1; y1=source.y1;
  x2=source.x2; y2=source.y2;

  // allocate memory
  val_array=source.val_array;
  iter_array=source.iter_array;
};



// setters
template<complexFunc_t F>
void Fractal<F>::Calculate(bool verbose) // serially loop through matrix entries
{
  for(int i=0; i<nRows; i++)
  {
    for(int j=0; j<nCols; j++)
    {
      if(verbose) printf("%d ", i*nCols+j);
      iter_array[i][j]=0;
      Complex c_init=val_array[i][j], z(0,0); // complex numbers c,z = a+ib
      int iter=1; // leave iter_array as 0 if recursion doesn't explode.
      do
      {
        z = (*F)(c_init,z); // assignment handled in my_recursive
        // if recursion > radius of fractal
        if(modulus2(z) >= radius*radius) 
        {
          val_array[i][j]=z; // copy z back to val_array
          iter_array[i][j]=iter; // update iter_array to nonzero interger
        }; 
        iter++;
      }
      while(modulus2(z) < radius*radius && iter<num_iters); // loop while inside radius and iterations remain
    }
    if(verbose) printf("\n");
  }
};

template<complexFunc_t F>
void Fractal<F>::CalculateCuda(bool verbose) // iterate on each matrix entry.
{
  // host memory 
  int num_elems = nRows*nCols;
  if(verbose) cout<<"host memory"<<endl;
  Complex* val_array_h = new Complex[num_elems];
  int* iter_array_h = new int[num_elems];
  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array_h[i*nCols+j]=val_array[i][j];
    }
  }

  // device memory 
  if(verbose) cout<<"device memory"<<endl;
  Complex* val_array_d;
  int* iter_array_d;
  hipMalloc(&val_array_d, num_elems*sizeof(Complex));
  hipMalloc(&iter_array_d, num_elems*sizeof(int));
  hipMemcpy(val_array_d, val_array_h, num_elems*sizeof(Complex), hipMemcpyHostToDevice);

  // run kernel
  if(verbose) if(verbose)cout<<"kernel running"<<endl;
  iteration_kernel<F><<<32,256>>>(iter_array_d, num_iters, radius, val_array_d, num_elems);
  hipDeviceSynchronize();

  //copy back to host buffer, unpack into val-array
  if(verbose) cout<<"copy back to host"<<endl;
  hipMemcpy(val_array_h, val_array_d, num_elems*sizeof(Complex), hipMemcpyDeviceToHost);
  hipMemcpy(iter_array_h, iter_array_d, num_elems*sizeof(int), hipMemcpyDeviceToHost);
  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array[i][j] = val_array_h[i*nCols+j];
      iter_array[i][j] = iter_array_h[i*nCols+j];
    }
  }

  // free memory
  if(verbose) cout<<"freeing memory"<<endl;
  delete[] val_array_h;
  delete[] iter_array_h;
  hipFree(val_array_d);
  hipFree(iter_array_d);
  
  // end of function
  if(verbose) cout<<"end of function"<<endl;
};

template<complexFunc_t F>
void Fractal<F>::CalculateCudaStreams(bool verbose) // iterate on each matrix entry.
{
  // host memory 
  int num_elems = nRows*nCols;
  if(verbose) cout<<"host memory"<<endl;
  Complex* val_array_h = new Complex[num_elems];
  int* iter_array_h = new int[num_elems];
  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array_h[i*nCols+j]=val_array[i][j];
    }
  }

  int num_gpus = 0;
  hipGetDeviceCount(&num_gpus);
  if(num_gpus==0){Fractal<F>::Calculate(verbose); return;};

  // device memory 
  if(verbose) cout<<"device memory"<<endl;
  Complex* val_array_d;
  int* iter_array_d;
  hipMalloc(&val_array_d, num_elems*sizeof(Complex));
  hipMalloc(&iter_array_d, num_elems*sizeof(int));

  int num_streams = 20;
  int stream_chunk_size = sdiv(num_elems,num_streams);

  hipStream_t streams_arr[num_streams];
  for(int i=0; i<num_streams; i++) hipStreamCreate(&streams_arr[i]);

  if(verbose) cout << "launching copy/compute in concurrent streams" << endl;
  for(int stream=0; stream<num_streams; stream++)
  {
    int stream_offset = stream*stream_chunk_size;
    int stream_num_elems = min(stream_chunk_size, num_elems-stream_offset);
    // copy compute copy back in stream
    hipMemcpyAsync(val_array_d+stream_offset,
                    val_array_h+stream_offset, 
                    stream_num_elems*sizeof(Complex),
                    hipMemcpyHostToDevice,
                    streams_arr[stream]);

    iteration_kernel<F><<<32,256,0,streams_arr[stream]>>>(iter_array_d+stream_offset, 
                                             num_iters, 
                                             radius, 
                                             val_array_d+stream_offset, 
                                             stream_num_elems);

    hipMemcpyAsync(val_array_h+stream_offset, 
                    val_array_d+stream_offset, 
                    stream_num_elems*sizeof(Complex), 
                    hipMemcpyDeviceToHost,
                    streams_arr[stream]);

    hipMemcpyAsync(iter_array_h+stream_offset, 
                    iter_array_d+stream_offset, 
                    stream_num_elems*sizeof(int), 
                    hipMemcpyDeviceToHost,
                    streams_arr[stream]);
  }
  
  hipDeviceSynchronize();
  for(int i=0; i<num_streams; i++) hipStreamDestroy(streams_arr[i]);

  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array[i][j] = val_array_h[i*nCols+j];
      iter_array[i][j] = iter_array_h[i*nCols+j];
    }
  }

  // free memory
  if(verbose) cout<<"freeing memory"<<endl;
  delete[] val_array_h;
  delete[] iter_array_h;
  hipFree(val_array_d);
  hipFree(iter_array_d);
  
  // end of function
  if(verbose) cout<<"end of function"<<endl;
};

template<complexFunc_t F>
void Fractal<F>::CalculateCudaGPUs(bool verbose) // iterate on each matrix entry.
{
  // host memory 
  int num_elems = nRows*nCols;
  if(verbose) cout<<"host memory"<<endl;
  Complex* val_array_h;
  int* iter_array_h;
  hipHostMalloc(&val_array_h, num_elems*sizeof(Complex));
  hipHostMalloc(&iter_array_h, num_elems*sizeof(int));
  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array_h[i*nCols+j]=val_array[i][j];
    }
  }

  int num_gpus = 0;
  hipGetDeviceCount(&num_gpus);
  if(num_gpus==0){cout<<"No GPUs found."<<endl; Fractal<F>::Calculate(verbose);return;};
  if(num_gpus==1){cout<<"1 GPU found."<<endl; Fractal<F>::CalculateCudaStreams(verbose); return;};
  int gpu_chunk_size = sdiv(num_elems, num_gpus);

  // device memory 
  if(verbose) cout<<"device memory"<<endl;
  Complex* val_array_d[num_gpus];
  int* iter_array_d[num_gpus];
  for(int gpu=0; gpu<num_gpus; gpu++)
  {
    hipSetDevice(gpu);
    int gpu_num_elems = min(gpu_chunk_size, num_elems-gpu*gpu_chunk_size);
    hipMalloc(&val_array_d[gpu], gpu_num_elems*sizeof(Complex));
    hipMalloc(&iter_array_d[gpu], gpu_num_elems*sizeof(int));
  }

  int num_streams = 20;
  int stream_chunk_size = sdiv(gpu_chunk_size,num_streams);

  hipStream_t streams_arr[num_gpus][num_streams];

  for(int i=0; i<num_gpus; i++) // initialize stream
  {
    hipSetDevice(i);
    for(int j=0; j<num_streams; j++) // init cuda streams
    {
      hipStreamCreate(&streams_arr[i][j]);
    };
  };

  if(verbose) cout << "launching copy/compute in concurrent streams on " << num_gpus << " GPUs." << endl;
  for(int gpu=0; gpu<num_gpus; gpu++)
  {
    hipSetDevice(gpu);
    int gpu_offset=gpu*gpu_chunk_size;
    int gpu_num_elems = min(gpu_chunk_size, num_elems-gpu_offset);

    for(int stream=0; stream<num_streams; stream++) // copy compute overlap
    {
      int stream_offset = stream*stream_chunk_size;
      int stream_num_elems = min(stream_chunk_size, gpu_num_elems-stream_offset);
      // copy compute copy back in stream
      hipMemcpyAsync(val_array_d[gpu]+stream_offset,
                      val_array_h+gpu_offset+stream_offset, 
                      stream_num_elems*sizeof(Complex),
                      hipMemcpyHostToDevice,
                      streams_arr[gpu][stream]);
  
      iteration_kernel<F><<<32,256,0,streams_arr[gpu][stream]>>>(iter_array_d[gpu]+stream_offset, 
                                              num_iters, 
                                              radius, 
                                              val_array_d[gpu]+stream_offset, 
                                              stream_num_elems);
      hipDeviceSynchronize();
      checkLastError();
  
      hipMemcpyAsync(val_array_h+gpu_offset+stream_offset, 
                      val_array_d[gpu]+stream_offset, 
                      stream_num_elems*sizeof(Complex), 
                      hipMemcpyDeviceToHost,
                      streams_arr[gpu][stream]);
        
      hipMemcpyAsync(iter_array_h+gpu_offset+stream_offset, 
                      iter_array_d[gpu]+stream_offset, 
                      stream_num_elems*sizeof(int), 
                      hipMemcpyDeviceToHost,
                      streams_arr[gpu][stream]);
      hipDeviceSynchronize();
      checkLastError();
    }
  }
  for(int gpu=0; gpu<num_gpus; gpu++) // synchronize all gpus
  {
    hipSetDevice(gpu);
    hipDeviceSynchronize();
  };

  for(int i=0; i<num_gpus; i++) // destroy all streams
  {
    for(int j=0; j<num_streams; j++)
    {
      hipSetDevice(i);
      hipStreamDestroy(streams_arr[i][j]);
    }
  }

  for(int i=0; i<nRows; i++) // copy val array into host buffer
  {
    for(int j=0; j<nCols; j++)
    {
      val_array[i][j] = val_array_h[i*nCols+j];
      iter_array[i][j] = iter_array_h[i*nCols+j];
    }
  }

  // free memory
  if(verbose) cout<<"freeing memory"<<endl;
  hipFree(val_array_h);
  hipFree(iter_array_h);
  for(int gpu=0; gpu<num_gpus; gpu++)
  {
    hipFree(val_array_d[gpu]);
    hipFree(iter_array_d[gpu]);
  }
  
  // end of function
  if(verbose) cout<<"end of function"<<endl;
};

template<complexFunc_t F>
void Fractal<F>::SetIterArr(vector<vector<int>> source_array) // set iter_array from another array
{
  nRows=source_array.size();
  nCols=source_array[0].size();
  iter_array=source_array;
};

template<complexFunc_t F>
void Fractal<F>::Center(Complex center_point) // move the mandelbrot set to be centered over a point p1
{
  double half_width = 0.5 * fabs(x2-x1);
  double half_height = 0.5 * fabs(y2-y1);

  Complex new_p1(center_point.real-half_width, center_point.im+half_height);
  Complex new_p2(center_point.real+half_width, center_point.im-half_height);

  x1=new_p1.real;
  y1=new_p1.im;
  x2=new_p2.real;
  y2=new_p2.im;

  // fill values
  for(int i=0; i<nRows; i++)
  {
    double y = y1 + (y2-y1)*(double(i)/(nRows-1));
    for(int j=0; j<nCols; j++)
    {
      double x = x1 + (x2-x1)*(double(j)/(nCols-1));
      val_array[i][j] = Complex(x,y);
    }
  }
}

template<complexFunc_t F>
void Fractal<F>::Zoom(double scale) // reset the coors (x1,y1), (x2,y2) to be smaller box around center
{
  double half_width = 0.5 * fabs(x2-x1);
  double half_height = 0.5 * fabs(y2-y1);
  Complex center_point((x2+x1)/2, (y2+y1)/2);

  scale = scale + (0.5 * (1-scale));
  Complex new_p1(center_point.real-half_width*scale, center_point.im+half_height*scale);
  Complex new_p2(center_point.real+half_width*scale, center_point.im-half_height*scale);


  x1=new_p1.real;
  y1=new_p1.im;
  x2=new_p2.real;
  y2=new_p2.im;

  // fill values
  for(int i=0; i<nRows; i++)
  {
    double y = y1 + (y2-y1)*(double(i)/(nRows-1));
    for(int j=0; j<nCols; j++)
    {
      double x = x1 + (x2-x1)*(double(j)/(nCols-1));
      val_array[i][j] = Complex(x,y);
    }
  }
};



// getters
template<complexFunc_t F>
vector<vector<Complex>> Fractal<F>::GetValArr() const// get iter_array from member data.
{
  return val_array;
};

template<complexFunc_t F>
vector<vector<int>> Fractal<F>::GetIterArr() const// get iter_array from member data.
{
  return iter_array;
};

template<complexFunc_t F>
void Fractal<F>::ValToCSV(string fname) const// write iters_array to a csv file
{
  ofstream out(fname);
  if(out.is_open())
  {
    for(int i=0; i<nRows; i++)
    {
      for(int j=0; j<nCols; j++)
      {
        out << "(" << val_array[i][j].real << ", " << val_array[i][j].im << "), ";
      }
      out << "\n";
    }
      out.close();
    cout << "array saved to " << fname << " successfully!" << endl;
  }
  else
  {
    cout << "failed to write file!" << endl;
  };
};

template<complexFunc_t F>
void Fractal<F>::ItersToCSV(string fname) const// write iters_array to a csv file
{
  ofstream out(fname);
  if(out.is_open())
  {
    for(int i=0; i<nRows; i++)
    {
      for(int j=0; j<nCols; j++)
      {
        out << iter_array[i][j] << ", ";
      }
      out << "\n";
    }
      out.close();
    cout << "array saved to " << fname << " successfully!" << endl;
  }
  else
  {
    cout << "failed to write file!" << endl;
  };
};

template<complexFunc_t F>
void Fractal<F>::ItersToIMG(string fname) const// write iters_array to a csv file
{
  ofstream out(fname);
  if(out.is_open())
  {
    // Write PPM header
    out << "P3\n";  // PPM magic number for color image (ASCII)
    out << nCols << " " << nRows << "\n";  // Image dimensions
    out << "255\n";  // Maximum color value (for grayscale, 255 is full white)
    for(int i=0; i<nRows; i++)
    {
      for(int j=0; j<nCols; j++)
      {
        if(iter_array[i][j]==1)
        {
          out << "255 190 190 "; // shade outside radius red
        }
        else if(iter_array[i][j]>1)
        {
          double val = iter_array[i][j];
          int red_cycle=21, blue_cycle=97, green_cycle=29;
          out << 255*fmod(val,red_cycle)/red_cycle << " " 
              << 255*fmod(val,blue_cycle)/blue_cycle << " " 
              << 255*fmod(val,green_cycle)/green_cycle << " ";  // RGB values are all the same for grayscale
        }
        else
        {
          out << "0 0 0 "; // color the mandelbrot set black
        }
      }
      out << "\n";
    }
      out.close();
    cout << "array saved to " << fname << " successfully!" << endl;
  }
  else
  {
    cout << "failed to write file!" << endl;
  };
};

template<complexFunc_t F>
cv::Mat Fractal<F>::ItersToFrame() const// write to a opencv cv::Mat
{
  cv::Mat frame(nRows, nCols, CV_8UC3);
  for(int i=0; i<nRows; i++)
  {
    for(int j=0; j<nCols; j++)
    {
      if(iter_array[i][j]==1)
      {
        frame.at<cv::Vec3b>(i,j) = cv::Vec3b(190,190,255); // shade outside radius red
      }
      else if(iter_array[i][j]>1)
      {
        double val = iter_array[i][j];
        int red_cycle=21, blue_cycle=97, green_cycle=29;
        frame.at<cv::Vec3b>(i,j) = cv::Vec3b(int(255*fmod(val,green_cycle)/green_cycle),
                                            int(255*fmod(val,blue_cycle)/blue_cycle),
                                             int(255*fmod(val,red_cycle)/red_cycle));
      }
      else
      {
        frame.at<cv::Vec3b>(i,j) = cv::Vec3b(0,0,0); // color the mandelbrot set black
      }
    }
  }
  if (frame.type() != CV_8UC3) {frame.convertTo(frame, CV_8UC3);}
  return frame;
};



// operators
template<complexFunc_t F>
Fractal<F>& Fractal<F>::operator = (const Fractal& source)
{
  if(this==&source) return *this;
  nRows=source.nRows;
  nCols=source.nCols;
  num_iters=source.num_iters;
  radius=source.radius;
  x1=source.x1;
  y1=source.y1;
  x2=source.x2;
  y2=source.y2;
  val_array=source.val_array; // array containing each pixel's value
  iter_array=source.iter_array; // array that counts how many iterations for a pixel to explode
  return *this;
}

#endif


